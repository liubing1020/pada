
#include <hip/hip_runtime.h>
#define RK(result, integVar, formula) \
	real __attribute__((unused)) xx_delta, F1, F2, F3, xx; \
	xx = integVar; \
	xx_delta = xx; F1 = (real)(formula); \
	xx_delta = xx + ((real)(1.0/2)) * F1 * ((real)DT); F2 = (real)(formula); \
	xx_delta = xx + ((real)(3.0/4)) * F2 * ((real)DT); F3 = (real)(formula); \
	result  = (((real)(2.0/9)) * F1 + ((real)(1.0/3)) * F2 + ((real)(4.0/9)) * F3 ) * ((real) DT);

