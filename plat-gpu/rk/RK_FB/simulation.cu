#include "hip/hip_runtime.h"
#include "api.cuh"
#include "config.cu"
#include "stdio.h"

#define VAR(nr, min, max, bins, initbin) \
        if (threadIdx.y == (nr * WORK / NR_VARS)) {\
                localData[NR_CONSTS + 2 * NR_VARS + nr] = localData[NR_CONSTS + nr] = min  + ((real)initbin + extractNumber()) * (max - min) / (real)bins; \
                x_bins[nr + NR_VARS * threadIdx.x] = initbin;\
        }

#define AVAR(nr, min, max, bins)

#define CONST(nr, min, max, bins) \
        if (threadIdx.y == (nr * WORK / NR_CONSTS)) {\
                localData[nr] = extractNumber() * (max - min) + min;\
        }

#define EQUATION(rest...)
#define AEQUATION(rest...)

__device__ void GenRandom(unsigned int *x_bins, real *buffer_base, int thread_step) {
        real *localData = buffer_base;
        #include "model.cu"
}


#undef VAR
#undef AVAR
#undef CONST
#undef EQUATION
#undef AEQUATION

#define OPEN
#define CLOSE
#define C(c)
#define K(n)
#define X(n)
#define AX(n)
#define XX
#define CONST(rest...)
#define VAR(rest...)
#define AVAR(rest...)
#define ZERO

#define EQUATION(nr, formula, list) formula + 2
#define AEQUATION(rest...)

const int total_weight = 0
#include "model.cu"
;

#undef OPEN
#undef CLOSE
#undef C
#undef K
#undef X
#undef AX
#undef XX
#undef EQUATION
#undef AEQUATION
#undef CONST
#undef VAR
#undef AVAR
#undef ZERO

#define OPEN
#define CLOSE
#define C(c)
#define K(n)
#define X(n)
#define AX(n)
#define XX
#define CONST(rest...)
#define VAR(rest...)
#define AVAR(rest...)
#define ZERO

#define EQUATION(nr, formula, list) const int weight_ ## nr = 0 formula + 2;
#define AEQUATION(rest...)

#include "model.cu"

#undef MUL
#undef DIV
#undef PLUS
#undef MINUS
#undef OPEN
#undef CLOSE
#undef C
#undef K
#undef X
#undef AX
#undef XX
#undef EQUATION
#undef AEQUATION
#undef CONST
#undef VAR
#undef AVAR
#undef ZERO


#define HALFDT ( DT/ (real)2)

__device__ void FatNodes(real * buffer_base, int thread_step) {
	real __attribute__((unused)) * localData = buffer_base;

#define CONST(nr, rest...)
#define VAR(nr, rest...) 
#define AVAR(nr, rest...) 
#define C(c) (real)c
#define K(n) localData[n]
#define X(n) localData[NR_CONSTS + 2 * NR_VARS + n]
#define AX(n) localData[NR_CONSTS + 2 * NR_VARS + n]
#define XX xx_delta
#define DIV /
#define MUL *
#define PLUS +
#define MINUS -
#define OPEN (
#define CLOSE )
#define ZERO 0

#define AEQUATION(nr, formula, list) if (threadIdx.y  == (nr - NR_VARS) * WORK / NR_AVARS) localData[NR_CONSTS + 2 * NR_VARS + nr] = (formula); 

#define EQUATION(nr, formula, list)

	#include "model.cu"


#undef X
#undef AX
#undef C
#undef K
#undef CONST
#undef VAR
#undef AVAR
#undef EQUATION
#undef AEQUATION
#undef DIV
#undef MUL
#undef PLUS
#undef MINUS
#undef OPEN
#undef CLOSE
#undef ZERO

}

__shared__ volatile int sync_work[WORK * ((PAR_SIMS + 31) / 32)];
#define SYNC_WORKERS {if (threadIdx.x % 32 == 0) {sync_work[threadIdx.y * ((PAR_SIMS + 31) / 32) + threadIdx.x / 32]++; \
                        int syncs; \
                        do { \
                                for (syncs = (WORK - 1) * ((PAR_SIMS + 31) / 32); syncs >= 0; syncs--) \
                                        if (sync_work[syncs] + 1 == sync_work[threadIdx.y * ((PAR_SIMS + 31) / 32) + threadIdx.x / 32]) break; \
                        } while (syncs >= 0);}}



__device__ void RungeKutta(real * buffer_base, int thread_step, unsigned int * isOK) {
	real * localData = buffer_base;

#define CONST(nr, rest...)
#define VAR(nr, rest...)
#define AVAR(nr, rest...)
#define C(c) (real)c
#define K(n) localData[n]
#define X(n) localData[NR_CONSTS + n]
#define XX xx_delta
#define DIV /
#define MUL *
#define PLUS +
#define MINUS -
#define OPEN (
#define CLOSE )
#define ZERO 0

/*
The distribution to several threads is done by comparing the current_weight to the total_weight
	This weight is determined based on the operators weight, statically (use compiler macro redefinition to list the operators weight only)
*/

#define EQUATION(nr, formula, list) \
	if (threadIdx.y  == (current_weight * WORK / total_weight)) {\
		RK(localData[NR_CONSTS + NR_VARS + nr], X(nr), formula, localData[NR_CONSTS + 3*NR_VARS + nr], isOK[nr]); \
	}\
	current_weight = current_weight + weight_ ## nr;

#define AEQUATION(nr, formula, list)

	int current_weight = 0;
	#include "model.cu"


#undef X
#undef C
#undef K
#undef CONST
#undef VAR
#undef AVAR
#undef EQUATION
#undef AEQUATION
#undef DIV
#undef MUL
#undef PLUS
#undef MINUS
#undef OPEN
#undef CLOSE
#undef ZERO
}

#define CONST(rest...) 




#define VAR(nr, min, max, bins, initbin) if (threadIdx.y == (WORK + (nr * ACCESS / (NR_VARS + NR_AVARS)))) { \
	bin = 0; \
	real val = localData[NR_CONSTS + 2 * NR_VARS + nr];\
	for (int i = 1; i < bins; i++) { \
		if (val >= min + ((real)((max - min ) * i)) / (bins > 0 ? bins : 1)) bin = i;\
	} \
	if (bins > 0) newx_bins[nr + (NR_VARS + NR_AVARS) * threadIdx.x] = bin;\
}

#define AVAR(nr, min, max, bins) if (threadIdx.y == (WORK + (nr * ACCESS / (NR_VARS + NR_AVARS)))) { \
	bin = 0; \
	real val = localData[NR_CONSTS + 2 * NR_VARS + nr];\
	for (int i = 1; i < bins; i++) { \
		if (val >= min + ((real)((max - min) * i)) / (bins > 0 ? bins : 1)) bin = i;\
	} \
	if (bins > 0) newx_bins[nr + (NR_VARS + NR_AVARS) * threadIdx.x] = bin;\
}

#define EQUATION(rest...) 
#define AEQUATION(rest...)

__device__ void BinPackVars(unsigned int *newx_bins, real *buffer_base, int thread_step) {
	real * localData = buffer_base;
	int bin;
	#include "model.cu"
}
#undef CONST
#undef VAR
#undef AVAR

#define CONST(nr, min, max, bins) \
	bin = 0; \
	for (int i = 1; i < bins; i++) { \
		if (localData[nr] >= min + ((real)((max - min) * i)) / (bins > 0 ? bins : 1)) bin = i;\
	} \
	if (bins > 0) k_bins[nr + threadIdx.x * NR_CONSTS] = bin;
         

#define VAR(rest...) 
#define AVAR(rest...) 

__device__ void BinPackConsts(unsigned int *k_bins, real *buffer_base, int thread_step) {
	int bin;
	real * localData = buffer_base;
	#include "model.cu"
}

#undef CONST
#undef VAR
#undef AVAR
#undef EQUATION
#undef AEQUATION

#define CONST(rest...)
#define VAR(rest...)
#define AVAR(rest...)

#define K(n) [k_bins[n + NR_CONSTS * threadIdx.x]]

#define X(n) [x_bins[n + NR_VARS * threadIdx.x]]

#define AX(n) [newx_bins[n + (NR_VARS + NR_AVARS) * threadIdx.x]]

#undef offsetof
#define offsetof(st, m) ((size_t) ( (unsigned int *)&((st *)(0))->m - (unsigned int *)0 ))

#define EQUATION(nr, formula, list) if (valid && (threadIdx.y == WORK + (nr * ACCESS / (NR_VARS + NR_AVARS)))) incBin(data_out, offsetof(struct xctr, x ## nr ## ctr [block] list [newx_bins[nr + (NR_VARS + NR_AVARS) * threadIdx.x]]));

#define AEQUATION(nr, formula, list) if (valid && (threadIdx.y == WORK + (nr * ACCESS / (NR_VARS + NR_AVARS)))) incBin(data_out, offsetof(struct xctr, x ## nr ## ctr [block] list [newx_bins[nr + (NR_VARS + NR_AVARS) * threadIdx.x]]));


__device__ void IncBins(unsigned int *k_bins, unsigned int *x_bins, unsigned int *newx_bins, void * data_out, unsigned int block, int valid) {

	#include "model.cu"

}

#undef CONST
#undef VAR
#undef AVAR
#undef K
#undef X
#undef AX
#undef EQUATION
#undef AEQUATION

__device__ int VOTE( unsigned int * isOK)
{
	for(int i = 0; i< NR_VARS; i++)
		{
			if(isOK[i] == 0) return 0;
		}
	return 1;
}

__device__ void INIT_ST(real * buffer_base, real val)
{
	for(int i = 0; i< NR_VARS; i++)
	buffer_base[NR_CONSTS + 3 * NR_VARS + i] = (real)val;
}

__device__ void SET_VOTE(real * buffer_base, unsigned int * isOK)
{
	real voted_st = buffer_base[NR_CONSTS + 3 * NR_VARS];
	for(int i = 0; i< NR_VARS; i++)
	{
		if(voted_st > buffer_base[NR_CONSTS + 3 * NR_VARS + i])
			voted_st = buffer_base[NR_CONSTS + 3 * NR_VARS+ i];
	}
	buffer_base[NR_CONSTS + 4 * NR_VARS] = voted_st;
	INIT_ST(buffer_base, voted_st);
}



__global__ void __launch_bounds__(PAR_SIMS * (WORK + ACCESS), 1) kernel(void *data_out, int traces, int seed, int thread_step) {
	__shared__ unsigned int *k_bins, *x_bins, *newx_bins, *checkST;
	extern  __shared__ real shared_mem[];
	if (threadIdx.x == 0 && threadIdx.y == 0) {
		checkST = (unsigned int *)malloc(NR_VARS * PAR_SIMS * sizeof(unsigned int));
		k_bins = (unsigned int *)malloc(NR_CONSTS * PAR_SIMS * sizeof(unsigned int));
		x_bins = (unsigned int *)malloc(NR_VARS * PAR_SIMS * sizeof(unsigned int));
		newx_bins = (unsigned int *)malloc((NR_VARS + NR_AVARS) * PAR_SIMS * sizeof(unsigned int));
	}
	initializeGenerator((seed * gridDim.x + blockIdx.x) * blockDim.y * blockDim.x + threadIdx.y * blockDim.x + threadIdx.x);
	__syncthreads();
	
	for (int i = blockIdx.x * PAR_SIMS + threadIdx.x; (i / PAR_SIMS) * PAR_SIMS < traces; i += PAR_SIMS * gridDim.x) {
		real *buffer_base = shared_mem + threadIdx.x * thread_step;
		unsigned int *isOK = checkST + threadIdx.x * NR_VARS;
		if (threadIdx.y < WORK) {
			GenRandom(x_bins, buffer_base, thread_step);	
		}
			// assign X and K for all vars, also the bins for X
		__syncthreads();
		if (threadIdx.y == 0) {
			BinPackConsts(k_bins, buffer_base, thread_step);
		}
		__syncthreads();

	
		
		for (int block = -1; block < BLOCKS; block++) {
			if ((threadIdx.y < WORK) && (block < BLOCKS - 1) && (threadIdx.x % 32 == 0)) 
				sync_work[threadIdx.y * ((PAR_SIMS + 31) / 32) + threadIdx.x / 32] = 0;
			__syncthreads();
			if (threadIdx.y < WORK) {
				if (block < BLOCKS - 1) {
							real time = 0.0;
							real time_step = 1E-3;
							real INTERVAL = STEPS * ((real)(time_step));	
							INIT_ST(buffer_base,time_step);
							while(time < INTERVAL)	{
									int lim = 0;
									do{
										RungeKutta(buffer_base, thread_step,isOK);  					
										SYNC_WORKERS
										++ lim;
									} while ((!VOTE(isOK)) && (lim < 10));
									SET_VOTE(buffer_base,isOK);
									RungeKutta(buffer_base, thread_step,isOK); 
									time += buffer_base[NR_CONSTS + 4 * NR_VARS];
									SYNC_WORKERS
									INIT_ST(buffer_base,buffer_base[NR_CONSTS + 4 * NR_VARS]);
									for (int j = threadIdx.y; j < NR_VARS; j+= WORK) 
									buffer_base[NR_CONSTS + j] += buffer_base[NR_CONSTS + NR_VARS + j];
									SYNC_WORKERS
							}
					}
			} else {
				if (block >= 0) {
					BinPackVars(newx_bins, buffer_base, thread_step);
					IncBins(k_bins, x_bins, newx_bins, data_out, block, i < traces);
					for (int j = threadIdx.y - WORK; j < NR_VARS; j+= ACCESS) 
						x_bins[j + NR_VARS * threadIdx.x] = newx_bins[j + (NR_VARS + NR_AVARS) * threadIdx.x];
				}
			}
			if (block < BLOCKS - 1 && threadIdx.y < WORK) FatNodes(buffer_base, thread_step);

			__syncthreads();
			if (block < BLOCKS - 1) for (int j = threadIdx.y; j < NR_VARS; j+= WORK + ACCESS) 
				buffer_base[NR_CONSTS + 2 * NR_VARS + j] = buffer_base[NR_CONSTS + j];
			__syncthreads();
		}
	}
}



#define COALESCE(n) ((n) % 2 == 0 ? (n) + 1 : (n))


void simulation(struct hipDeviceProp_t *props, void *data_out, int traces, unsigned int seed) {

	dim3 grid(props->multiProcessorCount);                                 // defines the number of blocks
        dim3 threads(PAR_SIMS, WORK + ACCESS);     // defines the pattern of threads inside each block
        int thread_step = COALESCE((NR_CONSTS + NR_VARS + NR_VARS + NR_VARS + NR_VARS + NR_AVARS + 1)); // coalescing is done at int level
        int sharedSize = PAR_SIMS * thread_step * sizeof(real);
        printf("Shared memory occupancy: %d (%.1f%%)\n", sharedSize, (float)sharedSize * 100 / props->sharedMemPerBlock);
		hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernel), hipFuncCachePreferShared);
		cudaThreadSetLimit(hipLimitMallocHeapSize, props->multiProcessorCount * 2 * PAR_SIMS * ((WORK + ACCESS) * 626 + NR_CONSTS + NR_VARS + NR_VARS + NR_VARS + NR_AVARS) * sizeof(unsigned int));
        kernel<<<grid, threads, sharedSize>>>(data_out, traces, seed, thread_step);
}


