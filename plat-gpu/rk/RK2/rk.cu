
#include <hip/hip_runtime.h>
#define RK(result, integVar, formula) \
	real xx, __attribute__((unused)) xx_delta, F1, F2;\
	xx = integVar;\
	xx_delta = xx;\
	F1 = (real)(formula);\
	xx_delta = xx + ((real)(2.0/3)) * F1 * ((real)DT);\
	F2 = (real)(formula);\
	result = (((real)(1.0/4)) * F1 + ((real)(3.0/4)) * F2 ) * ((real) DT);
