
#include <hip/hip_runtime.h>
// receives the offset in the table where the increment should take place //
// thread_id = [0, BIN_THREADS] is guaranteed to be different for all the threads that call incBin in parallel

__device__ void incBin(void *data_out, unsigned int offset, int thread_id) {
	unsigned int *where = ((unsigned int *)data_out) + offset;
	atomicAdd(where, 1);
}
