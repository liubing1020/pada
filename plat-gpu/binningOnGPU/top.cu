/**
   The top file of the simulation, loads the kernels after determining the runtime parameters
   The most important is derived as thread_step based on the number of constants and variables
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "multigpu.h"

// this is the prototype of a single GPU simulation
// the inner code does not make any assumptions on the data_out format, it carries the pointer onwards and passes it to the binning function
void simulation(struct hipDeviceProp_t *props, void *data_out, int traces, unsigned int seed);

int main(int argc, char **argv) {
	int fileid;
	if (argc < 2) {
		printf("%s\n", "Error: At least the number of traces needs to be specified");
		return -1;
	}
	int TRACES = atoi(argv[1]);
	if (TRACES < 1) {
		printf("%s\n", "Error: Invalid number of traces");
		return -1;
	}
	int WHICH_DEVICE = 0;
	if (argc > 2) { 
		if (strcmp(argv[2], "-gpu") == 0) {
			if (argc < 4) {
				printf("%s\n", "Error: Invalid GPU specified");
				return -1;
			}
			WHICH_DEVICE = atoi(argv[3]);
			if (hipSetDevice(WHICH_DEVICE) != hipSuccess) {
				printf("%s\n", "Error: Invalid GPU specified");
				return -1;
			}
			if (argc>=5){ //./exec 3000 -gpu 0 -master 4
				useMultiGPU=1;
				if(strcmp(argv[4], "-master") == 0) {
					MSstatus=MASTER;
					useMultiGPU=atoi(argv[5]);
					if(useMultiGPU > MAX_GPU){
						printf("Error: at most %d GPUs available\n", MAX_GPU);
						return -1;
					}
				}
				else if (strcmp(argv[4], "-slave")==0){
					MSstatus=SLAVE;
					fileid=atoi(argv[5]);
				}
			}
		}
	}

	struct xctr *bins_host;
	struct xctr *bins_gpu;
	hipEvent_t start, stop;
	bins_host = (struct xctr *)malloc(sizeof(struct xctr));
	if(!useMultiGPU || MSstatus==SLAVE){
		struct hipDeviceProp_t props;	
		hipGetDeviceProperties(&props, WHICH_DEVICE);

		hipMalloc((void **)&bins_gpu, sizeof(struct xctr));
		hipMemset(bins_gpu, 0, sizeof(struct xctr));

		hipDeviceSynchronize();
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord(start, 0);

		simulation(&props, bins_gpu, TRACES, WHICH_DEVICE);

		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		printf("Status: %s\n", hipGetErrorString(hipGetLastError()));
		hipMemcpy(bins_host, bins_gpu, sizeof(struct xctr), hipMemcpyDeviceToHost);
		hipDeviceSynchronize();
		float elapsedTime;
		hipEventElapsedTime(&elapsedTime, start, stop);
		printf("Time: %.2fs\n", (double)elapsedTime/1000);
		hipEventDestroy(start); 
		hipEventDestroy(stop);
		hipFree(bins_gpu);
	}

	//master thread will launch slave threads
	if(MSstatus==MASTER){
		printf("Using multi GPUs:\n");
		pthread_attr_t  attributes;
		pthread_attr_init(&attributes);
		pthread_attr_setdetachstate(&attributes, PTHREAD_CREATE_JOINABLE);

		pthread_t thread[MAX_GPU];
		ThreadData slavethd[MAX_GPU];
		ThreadData_master masterthd;

		//check gpus config
		if(gpudeviceinfo[0].device!=WHICH_DEVICE){ 
			printf("Please update multigpu.h\n");
			return -1;
		}
		masterthd._id=WHICH_DEVICE;
		masterthd._traces=TRACES/useMultiGPU;
		masterthd._data_h=bins_host;
		pthread_create(&thread[0], NULL, launchGPU_master, &masterthd);

		printf("master: device %d @ %s\n"
				,gpudeviceinfo[0].device, gpudeviceinfo[0].server);
		//create threads for slaves
		for(int i=1;i<useMultiGPU;i++){
			printf("slave: device %d @ %s\n"
					,gpudeviceinfo[i].device, gpudeviceinfo[i].server);
			slavethd[i]._id=i;
			slavethd[i]._MSstatus=SLAVE;
			//compare with master's server to check local/remote
			if(strcmp(gpudeviceinfo[i].server,gpudeviceinfo[0].server)==0){
				sprintf(slavethd[i]._cmd, "%s %d -gpu %d -slave %d"
						,argv[0], TRACES/useMultiGPU, gpudeviceinfo[i].device, i);
			}
			else{
				sprintf(slavethd[i]._cmd, "rsync -av %s %s@%s:%s/; ssh %s@%s 'cd %s; %s %d -gpu %d -slave %d'"
						, argv[0], USER, gpudeviceinfo[i].server, gpudeviceinfo[i].path
						, USER, gpudeviceinfo[i].server, gpudeviceinfo[i].path
						, argv[0], TRACES/useMultiGPU, gpudeviceinfo[i].device, i);
				sprintf(slavethd[i]._filecmd, "scp %s@%s:%s/bintemp%d ./"
						, USER, gpudeviceinfo[i].server, gpudeviceinfo[i].path, i);
			}
			pthread_create(&thread[i], NULL, launchGPU_slave, &slavethd[i]);
		}
		for(int i=0;i<useMultiGPU;i++){
			pthread_join(thread[i], NULL);
		}

		//collect output from slave gpus
		FILE *binpartfile;
		char binfilename[255];
		struct xctr *binpart;
		binpart=(struct xctr *)malloc(sizeof(struct xctr));
		//master gpu id is 0, the rest:     1,2,3 ...
		for(int i=1;i<useMultiGPU;i++){
			sprintf(binfilename,"bintemp%d",i);
			binpartfile=fopen(binfilename,"r");
			if(binpartfile!=NULL){
				fread(binpart,sizeof(struct xctr),1,binpartfile);
				fclose(binpartfile);
				unsigned int *u_part = ( unsigned int *)binpart;
				unsigned int *u_bin_host = ( unsigned int *)bins_host;
				for(int j=0;j<(sizeof(struct xctr)/sizeof(unsigned int));j++){
					u_bin_host[j]+=u_part[j];
				}
			}
		}
	}

	if(MSstatus==MASTER || !useMultiGPU){
		printf("Writing to files...\n");

		long checkup_counter = 0;

#include "WriteFiles.c"
		if (checkup_counter != (long)(NR_VARS + NR_AVARS) * BLOCKS * TRACES) printf("Checkup error on number of binned vars: %ld, should be %ld\n", checkup_counter, (long) (NR_VARS +NR_AVARS) * BLOCKS * TRACES);
		printf("Done\n");
	}
	else if (MSstatus==SLAVE){
		//slave gpu will write bin_host to a temp file
		FILE *tempfile;
		char tempfilename[255];
		sprintf(tempfilename, "bintemp%d", fileid);
		tempfile=fopen(tempfilename,"w");
		fwrite(bins_host,sizeof(struct xctr),1,tempfile);
		fclose(tempfile);
	}
	free(bins_host);

	return 0;
}
