#include "hip/hip_runtime.h"
__device__ void bin_pack(volatile struct struct_trace_data *this_trace, int nr, real val) {
        int bin = 0;
        for (int i = 1; i < var_bins[nr]; i++) {
                if (val >= var_min[nr] + ((real)((var_max[nr] - var_min[nr]) * i)) / var_bins[nr]) bin = i;
        }
        this_trace -> newx_bins[nr] = bin;
}

__device__ real get_random(real min, real max) {
        return extractNumber()  * (max - min) + min;
}

/* main entry point */
/* launch bounds to assist the compiler regarding the maximum number of registers per thread */

__global__ __launch_bounds__(nrWarps * 32, 1) void kernel(void * data_out, int traces, unsigned int seed) {
	__shared__ struct_trace_data *trace_data;
        volatile extern __shared__ real sharedData[];
	if (threadIdx.x == 0 && threadIdx.y == 0) {
		trace_data = (struct_trace_data *)malloc(32 * sizeof(struct_trace_data));
	}
	initializeGenerator((seed * gridDim.x + blockIdx.x) * blockDim.y * blockDim.x + threadIdx.y * blockDim.x + threadIdx.x);
        __syncthreads();

        /*   Each trace requires to have its working set in global memory;   **
        **   it will be prefetched as needed during computation              */

        volatile struct struct_trace_data *this_trace = &trace_data[threadIdx.x];

        for (int trace = blockIdx.x * 32 + threadIdx.x; (trace / 32) * 32 < traces ; trace += 32 * gridDim.x) {
                int i;

                if (threadIdx.y == 0) for (i = 0; i < NR_CONSTS; i++) {
			int bin = 0;
                        real val = get_random(const_min[i], const_max[i]);
                        this_trace -> traceConsts[i] = val;
                        for (int j = 1; j < const_bins[i]; j++) {
                                if (val >= const_min[i] + ((real)((const_max[i] - const_min[i]) * j)) / const_bins[i]) bin = j;
                        }
                        this_trace -> k_bins[i] = bin;
                }

                if (threadIdx.y == 0) for (i = 0; i < NR_VARS; i++) {
                        this_trace -> x_bins[i] = var_initbin[i];
                        this_trace -> traceVars[i] =
                                get_random(var_min[i] + (var_max[i] - var_min[i]) * (real)var_initbin[i] / (real)var_bins[i],
                                           var_min[i] + (var_max[i] - var_min[i]) * (real)(var_initbin[i] + 1) / (real)var_bins[i]);
                }

                __syncthreads();
		__threadfence();

                int block, t;

                for (block = 0; block < BLOCKS; block++) {
                        for (t = 0; t < STEPS; t++) {
                                compute(t, this_trace, sharedData + threadIdx.x * THREAD_STEP);
                                __syncthreads();
				__threadfence();
				for (i = threadIdx.y; i < NR_VARS; i += nrWarps) this_trace -> traceVars[i] = this_trace -> tracePost[i];
                                __syncthreads();
				__threadfence();
                	}



#undef offsetof
#define offsetof(st, m) ((size_t) ( (unsigned int *)&((st *)(0))->m - (unsigned int *)0 ))

#define BIN_THREADS (32 * gridDim.x * nrWarps)
#include "incBins.cu"

                        #define VAR(...)
                        #define AVAR(...)
                        #define CONST(...)
                        #define K(n) [this_trace -> k_bins[n]]
                        #define X(n) [this_trace -> x_bins[n]]
                        #define AX(n) [this_trace -> newx_bins[n]]
                        #define EQUATION(nr, formula, list) \
                                if (trace < traces && (nr % nrWarps == threadIdx.y)) incBin(data_out, offsetof(struct xctr, x ## nr ## ctr [block] list [this_trace -> newx_bins[nr]]), (blockIdx.x * 32 + threadIdx.x) * nrWarps + threadIdx.y);


                        #define AEQUATION(nr, formula, list) \
                                if (trace < traces && (nr % nrWarps == threadIdx.y)) incBin(data_out, offsetof(struct xctr, x ## nr ## ctr [block] list [this_trace -> newx_bins[nr]]), (blockIdx.x * 32 + threadIdx.x) * nrWarps + threadIdx.y);

                        #include "model.cu"

                        #undef X
                        #undef AX
                        #undef K
                        #undef EQUATION
                        #undef AEQUATION
                        #undef VAR
                        #undef AVAR
                        #undef CONST

                        __syncthreads();
			__threadfence();

                        for (i = threadIdx.y; i < NR_VARS; i += nrWarps) this_trace -> x_bins[i] = this_trace -> newx_bins[i];

                        __syncthreads();
			__threadfence();
                }
        }
}


void simulation(struct hipDeviceProp_t *props, void *data_out, int traces, unsigned int seed) {
	dim3 grid(props->multiProcessorCount);                                 // defines the number of blocks
        dim3 threads(32, nrWarps);     // defines the pattern of threads inside each block
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernel), hipFuncCachePreferShared);
	if (hipSuccess != cudaThreadSetLimit(hipLimitMallocHeapSize, props->multiProcessorCount * 8 * 32 * (nrWarps * 626 * sizeof(unsigned int) + sizeof(struct_trace_data)))) printf("Error allocating memory");
	hipDeviceSynchronize();
        kernel<<<grid, threads, 32 * THREAD_STEP * sizeof(real)>>>(data_out, traces, seed);
}


