
#include <hip/hip_runtime.h>
#define RK(result, integVar, formula) \
	real xx, __attribute__((unused)) xx_delta, halfF1, halfF2, F3, F4;\
	xx = integVar;\
	xx_delta = xx;\
	halfF1 = ((real)HALFDT) * (formula);\
	xx_delta = xx + halfF1;\
	halfF2 = ((real)HALFDT) * (formula);\
	xx_delta = xx + halfF2;\
	F3 = ((real)DT) * (formula);\
	xx_delta = xx + F3;\
	F4 = ((real)DT) * (formula);\
	result = (((real)2.0) * (halfF1 + F3) + (real)4.0 * halfF2 + F4) * ((real)1.0/(real)6.0);

