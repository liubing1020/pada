
#include <hip/hip_runtime.h>
#define RK(result, integVar, formula, new_step_size, isOK) \
		real __attribute__((unused)) xx_delta, F1, F2, F3, F4, F5, F6, init_value; \
		real __attribute__((unused)) err1, err3, err4, err5, err6, realerr,current_h, xx, REL_TOL, ABS_TOL, tol; \
		current_h = ((real) (new_step_size)); REL_TOL = (real)(1E-6); ABS_TOL = (real)(1E-6); \
		xx = integVar; xx_delta = xx; init_value = (real)(formula); \
		realerr = ((real)0.0); \
		xx_delta = xx; F1 =  current_h * ((real)(init_value)); \
		xx_delta = xx + ((real)1.0/4) * F1; F2 =  current_h *((real)(formula)); \
		xx_delta = xx + ((real)(3.0/32)) * F1 + ((real)(9.0/32)) * F2 ; F3 =  current_h * ((real)(formula)); \
		xx_delta = xx + ((real)(1932.0/2197)) * F1 + ((real)(-7200.0/2197)) * F2 + ((real)(7296.0/2197)) * F3 ; \
		F4 = current_h * (real)(formula); \
		xx_delta = xx + ((real)(439.0/216)) * F1 + ((real)(-8.0)) * F2 + ((real)(3680.0/513)) * F3 + ((real)(-845.0/4104)) * F4 ; \
		F5 =  current_h * (real)(formula);\
		xx_delta = xx + ((real)(-8.0/27)) * F1 + ((real)2.0) * F2 + ((real)(-3544.0/2565)) * F3 + ((real)(1859.0/4104)) * F4 + ((real)(-11/40)) * F5; \
		F6 = current_h * (real)(formula); \
		err1 = ((real)(1.0/360)) * F1; \
		err3 = ((real)(-128.0/4275)) * F3; \
		err4 = ((real)(-2197.0/75240)) * F4; \
		err5 = ((real)(1.0/50)) * F5; \
		err6 = ((real)(2.0/55)) * F6; \
		realerr = (real)(err1 + err3 + err4 + err5 + err6) ; \
		tol = max(fabs((((real)REL_TOL) * init_value)), ABS_TOL);\
		if (fabs(((real)realerr)) > tol) {\
		new_step_size = ((real)0.84) * ((real)current_h * pow(fabs( tol / ((real)realerr)), ((real)0.25))); isOK = 0;} \
		else {isOK = 1; }\
		result = ((real)(25.0/216)) * F1 + ((real)(1408.0/2565)) * F3 + ((real)(2197.0/4104)) * F4 + ((real)(-1.0/5)) * F5 ; \
 

