#include "hip/hip_runtime.h"
/*
(c) Andrei Hagiescu 2010
This file generates 'real' random numbers in a specified interval
It uses a Mersene RNG whose source code is specified on Wikipedia
*/

__shared__ unsigned int *indx, *MT;

__device__ void initializeGenerator(unsigned int which) {
	int threadId = threadIdx.y * blockDim.x + threadIdx.x;
	if (threadId == 0) {
		indx = (unsigned int *)malloc(blockDim.x * blockDim.y * sizeof(unsigned int));
		MT = (unsigned int *)malloc(blockDim.x * blockDim.y * 625 * sizeof(unsigned int));
	}
	__syncthreads();
	MT[threadId * 625] = which;
	for (int i = 1; i < 623; i++) MT[threadId * 625 + i] = 0x6c078965 * (MT[threadId * 625 + i-1] ^ (MT[threadId * 625 + i-1] >> 30)) + i;
	indx[threadId] = 0;
}
 

 // Generate an array of 624 untempered numbers
__device__ void generateNumbers() {
	int threadId = threadIdx.y * blockDim.x + threadIdx.x;
	for (int i = 0; i < 624; i++) {
		unsigned int y = (MT[threadId * 625 + i] & 0x01) + (MT[threadId * 625 + (i+1) % 624] & 0x7FFFFFFF);
		MT[threadId * 625 + i] = MT[threadId * 625 + (i + 397) % 624] ^ (y >> 1);
		if (y % 2 == 1) { // y is odd
			MT[threadId * 625 + i] = MT[threadId * 625 + i] ^ 0x9908b0df;
		}
	}
}


__device__ real extractNumber() {
	int threadId = threadIdx.y * blockDim.x + threadIdx.x;
	if (indx[threadId] == 0) generateNumbers();

	unsigned int y = MT[threadId * 625 + indx[threadId]];
	y = y ^ (y >> 11);
	y = y ^ ((y << 7) & 0x9d2c5680);
	y = y ^ ((y << 15) & 0xefc60000);
	y = y ^ (y >> 18);
	indx[threadId]++;
	if (indx[threadId] == 624) indx[threadId] = 0;
	return (real)y / (real)0xffffffff;
}

